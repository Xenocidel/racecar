#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "zedMagic.h"
#define square(x) x*x
#define THRESHOLD 70




__global__
void saxpy(int n, float a, float *x, float *y) {
    int i=blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) y[i]=a*x[i]+y[i];
}

extern "C" int testMain(void) {
    int N=30000;//1<<20;
    float *x, *y, *d_x, *d_y;
    
    x=(float*)malloc(N*sizeof(float));
    y=(float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i=0;i<N;i++) {
        x[i]=1.0f;
        y[i]=2.0f;  
    }

    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(float),hipMemcpyHostToDevice);


    saxpy<<<(N+255)/256, 256>>>(N,2.0f,d_x,d_y);

    hipMemcpy(y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);

    float maxError=0.0f;
    for (int i=0;i<N;i++) 
        maxError=max(maxError,abs(y[i]-4.0f));

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    return 0;
}


extern "C" std::vector<unsigned char> processImage(std::vector<unsigned char> imgData) {
    // Steps:
    // Read from std::vector format
    // Crop and scale
    // Grayscale    
    // Edge/SGM
    // Black and white
    // Return to std::vector format

    //Reading into an array AND cropping AND scaling at once
    unsigned char *scCrop,*d_scCrop;
    std::vector<unsigned char> output; 

    scCrop = (unsigned char*) malloc(3*H_O*W_O*sizeof(unsigned char));
    hipMalloc(&d_scCrop,3*H_O*W_O*sizeof(unsigned char));
    
    printf("allocated mem\n");
    if (scCrop==NULL) printf("Error allocating memory: allocated to NULL\n");

    int i,j;
    int W = W_I, H = H_I;
    int cropH=H*55/100;
    int newH=cropH/2;
    int newW=W/2;

    if (imgData.size() != W*H*3) {
        printf("ERROR dimensions wahwahwah\n");
    } 

    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O*3;j++) {
            long sum=0;
            sum=sum+imgData[(2*j)   + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j)   + 3*W*(2*i+1)];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i+1)];
            scCrop[i*W_O*3 + j] = (sum/4);
        }
    }

    printf("scaled and cropped into array\n");


    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O*3;j++) {
            output.push_back(scCrop[i*W_O*3 + j]);
        }
    }

    printf("read from array into vector\n");

    free(scCrop);
    hipFree(d_scCrop);

    std::vector <unsigned char> scaled; //to scale down image

    /*
    scaled = scaleAndCrop(imgData);
    * OR
    */

    scaled = output;
    
    std::vector <unsigned char> bw; // to hold raw bw
    
    bw = toGrayscale(scaled);

    
    std::vector <unsigned char> edges; // to hold scaled, thresholded sgm 
    edges = getEdges(bw);

    return edges;

}



