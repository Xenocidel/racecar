#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "zedMagic.h"
#define square(x) x*x
#define THRESHOLD 70




__global__
void saxpy(int n, float a, float *x, float *y) {
    int i=blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) y[i]=a*x[i]+y[i];
}

extern "C" int testMain(void) {
    int N=30000;//1<<20;
    float *x, *y, *d_x, *d_y;
    
    x=(float*)malloc(N*sizeof(float));
    y=(float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i=0;i<N;i++) {
        x[i]=1.0f;
        y[i]=2.0f;  
    }

    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(float),hipMemcpyHostToDevice);


    saxpy<<<(N+255)/256, 256>>>(N,2.0f,d_x,d_y);

    hipMemcpy(y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);

    float maxError=0.0f;
    for (int i=0;i<N;i++) 
        maxError=max(maxError,abs(y[i]-4.0f));

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    return 0;
}


extern "C" std::vector<unsigned char> processImage(std::vector<unsigned char> imgData) {
    // Steps:
    // Read from std::vector format
    // Crop and scale
    // Grayscale    
    // Edge/SGM
    // Black and white
    // Return to std::vector format

    //Reading into an array AND cropping AND scaling at once
    unsigned char *scCrop,*d_scCrop;
    std::vector<unsigned char> output; 

    scCrop = (unsigned char*) malloc(3*H_O*W_O*sizeof(unsigned char));
    hipMalloc(&d_scCrop,3*H_O*W_O*sizeof(unsigned char));
    
    printf("allocated mem\n");
    if (scCrop==NULL) printf("Error allocating memory: allocated to NULL\n");

    int i,j;
    int W = W_I, H = H_I;
    int cropH=H*55/100;
    int newH=cropH/2;
    int newW=W/2;

    if (imgData.size() != W*H*3) {
        printf("ERROR dimensions wahwahwah\n");
    } 

    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O*3;j++) {
            long sum=0;
            sum=sum+imgData[(2*j)   + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j)   + 3*W*(2*i+1)];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i+1)];
            scCrop[i*W_O*3 + j] = (sum/4);
        }
    }

    printf("scaled and cropped into array\n");


    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O*3;j++) {
            output.push_back(scCrop[i*W_O*3 + j]);
        }
    }

    printf("read from array into vector\n");

    free(scCrop);
    hipFree(d_scCrop);

    //return output;

   std::vector <unsigned char> scaled; //to scale down image

/*
    for (i=0;i<newH;i++) {
        for (j=0;j<newW*3;j++) {
            long sum=0;
            //sum += scaled[3*newW*i + 3*j +k];
            sum=sum+imgData[(2*j)   + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j)   + 3*W*(2*i+1)];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i+1)];
            scaled.push_back(sum/4);
        }
    }
*/

    scaled = output;
    
    std::vector <unsigned char> bw; // to hold raw bw
    
    /* Consolidate color into black and white photo */
    for (i=0;i<newH;i++) {
        for (j=0;j<newW;j++) {
            long sum = 0;
            int k;
            for (k=-1;k<2;k++) sum += scaled[3*newW*i + 3*j +k];
            bw.push_back(sum/3);
        }
    }
    
    std::vector <double> edges; // to hold sgm 
    /* Calculate dx, dy, sgm */
    int dx,dy;
    double max = 0;
    for (i=0;i<newH;i++) {
        if (i==0 || i==(newH-1)) { // top/bottom row
            for (j=0;j<newW;j++) edges.push_back(0);
        }
        else {
            edges.push_back(0); // left column
            for (j=1;j<newW-1;j++) {
                int index = newW*i + j; 
                dx = bw[index+newW+1] + 2*bw[index+1] + bw[index-newW+1] \
                - (bw[index+newW-1] + 2*bw[index-1] + bw[index-newW-1]);
                dy = bw[index-newW-1] + 2*bw[index-newW] + bw[index-newW+1]\
                - (bw[index+newW-1] + 2*bw[index+newW] + bw[index+newW+1]);
                double sgm = sqrt(square(dx) + square(dy));
                max = (max<sgm)? sgm: max;
                edges.push_back(sgm);
            }
            edges.push_back(0); // right column
        }
    }
    bw.clear();

    std::vector <unsigned char> edge_img;
    /* Scale edge (sgm) data to 0 to 255 range */
    for (i=0;i<edges.size();i++) edge_img.push_back((edges[i]/max*255 > THRESHOLD)?255 :0);
    edges.clear();

    return edge_img;


}



