#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "zedMagic.h"
#define square(x) x*x
#define THRESHOLD 70




__global__
void saxpy(int n, float a, float *x, float *y) {
    int i=blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) y[i]=a*x[i]+y[i];
}

__global__
void edgeMath(unsigned char* bw, float* edges) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    if (i==0 || i==H_O-1) edges[index] = 0;
    else {
        if (j==0 || j==W_O-1) edges[index] = 0;
        else {
            int dx,dy;
            int newW = W_O;
                dx = bw[index+newW+1] + 2*bw[index+1] + bw[index-newW+1] \
                - (bw[index+newW-1] + 2*bw[index-1] + bw[index-newW-1]);
                dy = bw[index-newW-1] + 2*bw[index-newW] + bw[index-newW+1]\
                - (bw[index+newW-1] + 2*bw[index+newW] + bw[index+newW+1]);
                double sgm = (square(dx) + square(dy));
            edges[index] = sgm;
        }
    }
}



extern "C" int testMain(void) {
    int N=30000;//1<<20;
    float *x, *y, *d_x, *d_y;
    
    x=(float*)malloc(N*sizeof(float));
    y=(float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i=0;i<N;i++) {
        x[i]=1.0f;
        y[i]=2.0f;  
    }

    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(float),hipMemcpyHostToDevice);


    saxpy<<<(N+255)/256, 256>>>(N,2.0f,d_x,d_y);

    hipMemcpy(y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);

    float maxError=0.0f;
    for (int i=0;i<N;i++) 
        maxError=max(maxError,abs(y[i]-4.0f));

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    return 0;
}


extern "C" std::vector<unsigned char> processImage(std::vector<unsigned char> imgData) {
    // Steps:
    // Read from std::vector format
    // Crop and scale
    // Grayscale    
    // Edge/SGM
    // Black and white
    // Return to std::vector format

    //Reading into an array AND cropping AND scaling at once
    unsigned char *scCrop,*d_scCrop;
    float  *edge,*d_edge;
    std::vector<unsigned char> output; 

    scCrop = (unsigned char*) malloc(H_O*W_O*sizeof(unsigned char));
    edge = (float*) malloc(H_O*W_O*sizeof(float));
    hipMalloc(&d_scCrop,H_O*W_O*sizeof(unsigned char));
    hipMalloc(&d_edge,H_O*W_O*sizeof(float));
    

    printf("allocated mem\n");
    if (scCrop==NULL) printf("Error allocating memory: allocated to NULL\n");

    int i,j;
    int W = W_I, H = H_I;

    if (imgData.size() != W*H*3) {
        printf("ERROR dimensions wahwahwah\n");
    } 

    /*for (i=0;i<H_O;i++) {
        for (j=0;j<W_O*3;j++) {
            long sum=0;
            sum=sum+imgData[(2*j)   + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i)  ];
            sum=sum+imgData[(2*j)   + 3*W*(2*i+1)];
            sum=sum+imgData[(2*j+3) + 3*W*(2*i+1)];
            scCrop[i*W_O*3 + j] = (sum/4);
        }
    }*/

    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O;j++) {
            long sum = 0;
            for (int k = 0; k<6; k++) {
                sum += imgData[6*j + k + 6*W*i];
                sum += imgData[6*j + k + 6*W*i + 3*W];
            }
            scCrop[i*W_O + j] = sum/12;
        }
    }

    printf("scaled and cropped into array\n");

    /*
    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O*3;j++) {
            output.push_back(scCrop[i*W_O*3 + j]);
        }
    }*/
    hipMemcpy(d_scCrop,scCrop,H_O*W_O*sizeof(unsigned char),hipMemcpyHostToDevice);
    edgeMath<<<W_O,H_O>>>(d_scCrop,d_edge);
    hipMemcpy(edge,d_edge,H_O*W_O*sizeof(float),hipMemcpyDeviceToHost);

    double max;
    for (i=0;i<H_O;i++) for (j=0;j<W_O;j++) max = (sqrt(edge[i*W_O + j])>max)? sqrt(edge[i*W_O + j]) : max;
 
    for (i=0;i<H_O;i++) for (j=0;j<W_O;j++) output.push_back( (sqrt(edge[i*W_O + j])/max*255 > THRESHOLD)? 255: 0);

    printf("read from array into vector\n");

    free(scCrop);
    hipFree(d_scCrop);
    free(edge);
    hipFree(d_edge);

    //output = toGrayscale(output); 
    //output = getEdges(output);
    return output;

}



