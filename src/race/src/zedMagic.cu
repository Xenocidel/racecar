#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "zedMagic.h"
#define square(x) x*x
#define THRESHOLD 70

#define AREA H_O*W_O


__global__
void saxpy(int n, float a, float *x, float *y) {
    int i=blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) y[i]=a*x[i]+y[i];
}

__global__
void edgeMath(unsigned char* bw, float* edges) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    if (i==0 || i==H_O-1) edges[index] = 0;
    else {
        if (j==0 || j==W_O-1) edges[index] = 0;
        else {
            int dx,dy;
            int newW = W_O;
                dx = bw[index+newW+1] + 2*bw[index+1] + bw[index-newW+1] \
                - (bw[index+newW-1] + 2*bw[index-1] + bw[index-newW-1]);
                dy = bw[index-newW-1] + 2*bw[index-newW] + bw[index-newW+1]\
                - (bw[index+newW-1] + 2*bw[index+newW] + bw[index+newW+1]);
                double sgm = (square(dx) + square(dy));
            edges[index] = sgm;
        }
    }
}



extern "C" int testMain(void) {
    int N=30000;//1<<20;
    float *x, *y, *d_x, *d_y;
    
    x=(float*)malloc(N*sizeof(float));
    y=(float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i=0;i<N;i++) {
        x[i]=1.0f;
        y[i]=2.0f;  
    }

    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(float),hipMemcpyHostToDevice);


    saxpy<<<(N+255)/256, 256>>>(N,2.0f,d_x,d_y);

    hipMemcpy(y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);

    float maxError=0.0f;
    for (int i=0;i<N;i++) 
        maxError=max(maxError,abs(y[i]-4.0f));

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    return 0;
}

extern "C" std::vector<unsigned char> processImage(std::vector<unsigned char> imgData, int func_id) {
    switch(func_id) {
        case CUDA_BASIC: return processImageCuda(imgData);
        case DUMB: return processImageDumb(imgData);
        case DUMB_MIN: return processImageDumb(imgData);
    } return processImageCuda(imgData);
} 


extern "C" std::vector<unsigned char> processImageCuda(std::vector<unsigned char> imgData) {
    // Steps:
    // Read from std::vector format
    // Crop and scale
    // Grayscale    
    // Edge/SGM
    // Black and white
    // Return to std::vector format

    //Reading into an array AND cropping AND scaling at once
    unsigned char *scCrop,*d_scCrop;
    float  *edge,*d_edge;
    std::vector<unsigned char> output; 

    scCrop = (unsigned char*) malloc(AREA*sizeof(unsigned char));
    edge = (float*) malloc(AREA*sizeof(float));
    hipMalloc(&d_scCrop,AREA*sizeof(unsigned char));
    hipMalloc(&d_edge,AREA*sizeof(float));
    
    int i,j;
    int W = W_I, H = H_I;

    if (imgData.size() != W*H*3) {
        printf("ERROR dimensions wahwahwah\n");
    } 

    for (i=0;i<H_O;i++) {
        for (j=0;j<W_O;j++) {
            long sum = 0;
            for (int k = 0; k<6; k++) {
                sum += imgData[6*j + k + 6*W*i];
                sum += imgData[6*j + k + 6*W*i + 3*W];
            }
            scCrop[i*W_O + j] = sum/12;
        }
    } //scaled and cropped into array

    hipMemcpy(d_scCrop,scCrop,AREA*sizeof(unsigned char),hipMemcpyHostToDevice);
    edgeMath<<<W_O,H_O>>>(d_scCrop,d_edge);
    hipMemcpy(edge,d_edge,AREA*sizeof(float),hipMemcpyDeviceToHost);

    double max;
    for (i=0;i<H_O;i++) for (j=0;j<W_O;j++) max = (sqrt(edge[i*W_O + j])>max)? sqrt(edge[i*W_O + j]) : max;
 
    for (i=0;i<H_O;i++) for (j=0;j<W_O;j++) output.push_back( (sqrt(edge[i*W_O + j])/max*255 > THRESHOLD)? 255: 0);


    free(scCrop);
    hipFree(d_scCrop);
    free(edge);
    hipFree(d_edge);

    return output;

}

extern "C" std::vector<unsigned char> processImageDumb(std::vector<unsigned char> imgData) {
    std::vector <unsigned char> output;
    output = scaleAndCrop(imgData);
    output = toGrayscale(output);
    output = getEdges(output);
    return output;
}

extern "C" std::vector<unsigned char> processImageMin(std::vector<unsigned char> imgData) {
    std::vector <unsigned char> output;
    output = scaleCropGrayscale(imgData);
    output = getEdges(output);
    return output;
}




#ifdef INPROGRESS
__global__
void findHough(unsigned char* edge, unsigned char* temp) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = index/W_O;
    int j = index%W_O;
    int x = j;
    int y = newH -1 -i;
    if (edge[index]) {
                
    }

}


void blah() {
    
  for (i=0;i<newH;i++) {
        for (j=0;j<newW;j++) {
            int x,y;
            x = j;
            y = newH - 1 - i;
            int index = newW*i + j;
            if (edge_img[index]) {
                for (theta=0;theta<180;theta++) {
                    roh = -x*sin(theta*M_PI/180) + y*cos(theta*M_PI/180);
                    roh_min = (roh_min>roh)? roh: roh_min;
                    roh_max = (roh_max<roh)? roh: roh_max;
                    voting[theta].push_back(roh);    
                }
            } 
        }
    }

    int hough_max = 0;
    int final_r, final_t;
    for (theta=0;theta<180;theta++) {
        for (roh=roh_min;roh<roh_max+1;roh++) {
            if (count(voting,theta,roh)>hough_max) {
                hough_max = count(voting,theta,roh);
                final_r = roh;
                final_t = theta;
            }
        }
    }

    printf("r %d, t %d\n", roh, theta);

    for (i=0;i<newH;i++) {
        for (j=0;j<newW;j++) {
            int x = j;
            int y = newH-1-i;
            if (abs(x*sin(final_t*M_PI/180) - y*cos(final_t*M_PI/180) + final_r) < 1) edge.data.push_back(255);
            else edge.data.push_back(0);
        }
    }


}

#endif
